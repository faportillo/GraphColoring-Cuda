#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <set>
#include <sstream>
#include <string>
#include <fstream>
#include <iostream>
#include <cstring>
#include <math.h>
#include <thrust/count.h>
#include <thrust/logical.h>
#include <thrust/functional.h>

#define MAXBLOCKS 1<<30

using namespace std;

//find min color
__device__
int min_color(int v,int n,int* colorMask){
	int i = 1;
	while((colorMask[i] == v) && (i < n)){
		i++;
	}

	return i;
}

__global__
void colorTopoKernel(int n, int* NNZ, int* preSum, int* colIndex, int* colors, bool* changed, bool* colored){

    int index = blockIdx.x * blockDim.x + threadIdx.x;
	int stride = blockDim.x*gridDim.x;
    int c;
    int colorMask[256];
    for (int i = 0; i < 256; i++){
    	colorMask[i] = 512;
    }

	for(int i = index; i < n; i+= stride){

        if (!colored[i]){
        	for (int k = preSum[i]; k < preSum[i + 1]; k++){
			    int j = colIndex[k];
			    colorMask[colors[j]] = i;
			}
			c = min_color(i, n, colorMask);
            colors[i] = c;
            colored[i] = true;
            changed[i] = true;
        }
	}
}

//check for collisions due to coloring adjacent nodes same color
__global__
void checkCollisions(int n, int* NNZ, int* preSum, int* colIndex, int* colors, bool* colored){
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	int stride = blockDim.x*gridDim.x;

	for(int i = index; i < n; i+=stride){
		for (int k = preSum[i]; k < preSum[i + 1]; k++){
		    int j = colIndex[k];
		    if ((colors[i] == colors[j]) && ( i < j)){
		    	colored[i] = false;
		    }
		}
	}
}

//topology driven parallel graph coloring algorithm
void colorTopo(int n, int* NNZ, int* preSum, int* colIndex, int* colors){
	bool* changed;
    bool* colored;

    hipMallocManaged(&colored, sizeof(bool)*n);
    hipMallocManaged(&changed, sizeof(bool)*n);

    thrust::fill(colored, colored+n, false);
    thrust::fill(colors, colors+n,0);

    do{
    	thrust::fill(changed, changed+n, false);
    	int nt = 256;
    	int nb = min((n + nt -1)/nt, MAXBLOCKS);
    	colorTopoKernel<<<nb,nt>>>(n, NNZ, preSum, colIndex, colors, changed, colored);
    	hipDeviceSynchronize();
    	checkCollisions<<<nb,nt>>>(n, NNZ,preSum, colIndex,  colors,colored);
    	hipDeviceSynchronize();
    }while(thrust::any_of(changed,changed+n,thrust::identity<bool>()));
    
    hipFree(colored);
    hipFree(changed);
}

//jones plassmann luby algorithm for graphcoloring
__global__ 
void colorJPLKernel(int n, int c, int* NNZ, int* preSum, int* colIndex,int* randoms, int* colors){

	int index = blockIdx.x * blockDim.x + threadIdx.x;
	int stride = blockDim.x*gridDim.x;
    
    if (index < n){
    	for (int i = index; i < n; i += stride){
		bool f = true;

		if ((colors[i] != -1 )){
			continue;
			
		}

		int ir = randoms[i];
        //instead of looping through 
		for (int k = preSum[i]; k < preSum[i + 1]; k++){
			int j = colIndex[k];
			int jc = colors[j];
			if (((jc != -1) && (jc != c)) || (i == j)){
				continue;
			}
			int jr = randoms[j];
			if (ir <= jr){
				f = false;
			}
		}
		if (f){
			colors[i] = c;
		}
	}
    }
	
}

//jones plassmann luby algorithm for graphcoloring
void colorJPL(int n, int* NNZ, int* preSum, int* colIndex, int* colors){
    int* randoms;
    hipMallocManaged(&randoms, sizeof(int)* n);
    for (int i = 0; i  < n; i++){
    	randoms[i] = rand(); 
    }

    thrust::fill(colors, colors+n, -1);

    for(int c = 1; c< n+1; c++){
    	int nt = 256;
    	int nb = min((n + nt -1)/nt, MAXBLOCKS);

    	colorJPLKernel<<<nb,nt>>>(n,c,NNZ,preSum,colIndex, randoms,colors);
        hipDeviceSynchronize();
    	int left = (int)thrust::count(colors, colors+n, -1);
    	if (left ==0){
    		break;
    	}
    }

    hipFree(randoms);
}

// Counts the number of unique colors in a solution
int CountColors(int V, int* color){
   int num_colors = 0;
   set<int> seen_colors;

   for (int i = 0; i < V; i++) {
      if (seen_colors.find(color[i]) == seen_colors.end()) {
         seen_colors.insert(color[i]);
         num_colors++;
      }  
   }

   return num_colors;
}

// Returns true if the color assignment is valid for the graph
bool IsValidColoring(bool* graph, int V, int* color) {
   for (int i = 0; i < V; i++) {
      for (int j = 0; j < V; j++) {
         if (graph[i * V + j]) {
            if (i != j && color[i] == color[j]) {
               printf("Vertex %d and Vertex %d are connected and have the same color %d\n", i, j, color[i]);
               return false;
            }
            if (color[i] < 1) {
               printf("Vertex %d has invalid color %d\n", i, color[i]);
               return false;
            }
         }
      }
   }

   return true;
}

// Read DIMACS graphs
// Assumes input nodes are numbered starting from 1
void ReadColFile(const char filename[], bool** graph, int* V) {
   string line;
   ifstream infile(filename);
   if (infile.fail()) {
      printf("Failed to open %s\n", filename);
      return;
   }

   int num_rows, num_edges;

   while (getline(infile, line)) {
      istringstream iss(line);
      string s;
      int node1, node2;
      iss >> s;
      if (s == "p") {
         iss >> s; // read string "edge"
         iss >> num_rows;
         iss >> num_edges;
         *V = num_rows;
         *graph = new bool[num_rows * num_rows];
         memset(*graph, 0, num_rows * num_rows * sizeof(bool));
         continue;
      } else if (s != "e")
         continue;
      
      iss >> node1 >> node2;

      // Assume node numbering starts at 1
      (*graph)[(node1 - 1) * num_rows + (node2 - 1)] = true;
      (*graph)[(node2 - 1) * num_rows + (node1 - 1)] = true;
   }
   infile.close();
}

// Read MatrixMarket graphs
// Assumes input nodes are numbered starting from 1
void ReadMMFile(const char filename[], bool** graph, int* V) {
   string line;
   ifstream infile(filename);
   if (infile.fail()) {
      printf("Failed to open %s\n", filename);
      return;
   }

   // Reading comments
   while (getline(infile, line)) {          
      istringstream iss(line);
      if (line.find("%") == string::npos)
         break;
   }

   // Reading metadata
   istringstream iss(line);
   int num_rows, num_cols, num_edges;
   iss >> num_rows >> num_cols >> num_edges;

   *graph = new bool[num_rows * num_rows];
   memset(*graph, 0, num_rows * num_rows * sizeof(bool));
   *V = num_rows;

   // Reading nodes
   while (getline(infile, line)) {          
      istringstream iss(line);
      int node1, node2, weight;
      iss >> node1 >> node2 >> weight;
      
      // Assume node numbering starts at 1
      (*graph)[(node1 - 1) * num_rows + (node2 - 1)] = true;
      (*graph)[(node2 - 1) * num_rows + (node1 - 1)] = true;
   }
   infile.close();
}

//store sparse graph in compressed sparse row format
void CSRConvert(bool** graph, int rows, int** NNZ, int* preSum, int** colIndex, int* counter){

    //assume square matrix
    int cols = rows;
    *counter = 0;
    int rowElem[rows];

    for (int i = 0; i < rows; i++){
        rowElem[i] = 0;
    }
    //initialize preSum
    preSum[0] = 0;

    for (int i = 0; i < rows; i++){
        for (int j = 0; j < cols; j++){
            if ((*graph)[i*rows + j] == false){
                continue;
            }
            else{
                //reallocate size of NNZ and colIndex
               *NNZ = (int*)realloc(*NNZ, sizeof(int)*(*counter + 1));
                 
                (*NNZ)[*counter] = 1;

                *colIndex = (int*)realloc(*colIndex, sizeof(int) * (*counter +1));
                (*colIndex)[*counter] = j;

                //preSum[counter + 1] = preSum[counter] + prevRowCount;
                rowElem[i]++;
                *counter += 1;
            }
        }
    }

    for (int i = 0; i < rows +1; i++){
        preSum[i+1] = preSum[i] + rowElem[i];
    }
}

//Assignment API
void GraphColoringGPU(const char filename[], int** color){
    bool* graph;
    int V;

    if (string(filename).find(".col") != string::npos)
      ReadColFile(filename, &graph, &V);
    else{
      ReadMMFile(filename, &graph, &V);
    }

    //convert the sparse array into compact sparse row format
    int *NNZ = (int*)malloc(sizeof(int)); 
    int *preSum = (int*)malloc(sizeof(int) * (V + 1));
    int *colIndex= (int*)malloc(sizeof(int));
    int counter = 0;
    CSRConvert(&graph, V, &NNZ, preSum, &colIndex, &counter);
    
    //migrate values to GPU
    int* Ao;
    int* Av;
    int* Ac;
    int* colors;
    hipMallocManaged(&Ao, sizeof(int)*(V+1));
    hipMallocManaged(&Av, sizeof(int)*counter);
    hipMallocManaged(&Ac, sizeof(int)*counter);
    hipMallocManaged(&colors, sizeof(int)* V);
    
    
    for(int i = 0; i < counter; i++){
    	Av[i] = NNZ[i];
    	Ac[i] = colIndex[i];
    }
    
    
    //printf("offset values : ");
    for (int i = 0; i < V + 1; i++){
    	Ao[i] = preSum[i];

    }
    
    // colorJPL(V, Av, Ao, Ac,colors);
    // printf("JPL coloring found solution with %d colors\n", CountColors(V, colors));
    // printf("Valid coloring: %d\n", IsValidColoring(graph, V, colors));

    colorTopo(V,Av,Ao,Ac,colors);

    printf("Topo coloring found solution with %d colors\n", CountColors(V, colors));
    printf("Valid coloring: %d\n", IsValidColoring(graph, V, colors));
    
    free(NNZ);
    free(preSum);
    free(colIndex);
    hipFree(Ao);
    hipFree(Av);
    hipFree(Ac);

}
