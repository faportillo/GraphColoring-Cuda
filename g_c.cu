#include "hip/hip_runtime.h"
/*
  Felix A. Portillo
  EEC289Q HW3
*/

#include <iostream>
#include <math.h>
#include <stdlib.h>
#include <stdio.h>
#include <set>
#include <sstream>
#include <string>
#include <fstream>
#include <iostream>
#include <cstring>

//Can i just do <thrust> ????
#include <thrust/count.h>
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>

using namespace std;

void GraphColoringGPU(const char filename[], int** color);
// Read MatrixMarket graphs
// Assumes input nodes are numbered starting from 1
void ReadMMFile(const char filename[], bool** graph, int* V) 
{
   string line;
   ifstream infile(filename);
   if (infile.fail()) {
      printf("Failed to open %s\n", filename);
      return;
   }

   // Reading comments
   while (getline(infile, line)) {          
      istringstream iss(line);
      if (line.find("%") == string::npos)
         break;
   }

   // Reading metadata
   istringstream iss(line);
   int num_rows, num_cols, num_edges;
   iss >> num_rows >> num_cols >> num_edges;

   *graph = new bool[num_rows * num_rows];
   memset(*graph, 0, num_rows * num_rows * sizeof(bool));
   *V = num_rows;

   // Reading nodes
   while (getline(infile, line)) {          
      istringstream iss(line);
      int node1, node2, weight;
      iss >> node1 >> node2 >> weight;
      
      // Assume node numbering starts at 1
      (*graph)[(node1 - 1) * num_rows + (node2 - 1)] = true;
      (*graph)[(node2 - 1) * num_rows + (node1 - 1)] = true;
   }
   infile.close();
}


// Read DIMACS graphs
// Assumes input nodes are numbered starting from 1
void ReadColFile(const char filename[], bool** graph, int* V) 
{
   string line;
   ifstream infile(filename);
   if (infile.fail()) {
      printf("Failed to open %s\n", filename);
      return;
   }

   int num_rows, num_edges;

   while (getline(infile, line)) {
      istringstream iss(line);
      string s;
      int node1, node2;
      iss >> s;
      if (s == "p") {
         iss >> s; // read string "edge"
         iss >> num_rows;
         iss >> num_edges;
         *V = num_rows;
         *graph = new bool[num_rows * num_rows];
         memset(*graph, 0, num_rows * num_rows * sizeof(bool));
         continue;
      } else if (s != "e")
         continue;
      
      iss >> node1 >> node2;

      // Assume node numbering starts at 1
      (*graph)[(node1 - 1) * num_rows + (node2 - 1)] = true;
      (*graph)[(node2 - 1) * num_rows + (node1 - 1)] = true;
      
   }
   //Print input matrix
   for (int i=0; i<num_rows; i++){
      for(int j=0; j<num_rows; j++){
        cout << (*graph)[i+num_rows+j]<<"   ";
      }
      cout << "\n\n";
   }
   infile.close();
}  

/*
  API deliverable. Put code here instead of in main!!
*/
void GraphColoringGPU(const char filename[], int** color){
  bool* graph;
  int V; //num rows
  int* color;
  
  //Experiment with thrust vectors
  thrust::device_vector<bool> dev_graph(graph,graph+V*V);
  int deg = thrust::count(thrust::device, dev_graph.begin(),dev_graph.end(),1);
  
   
   if (string(filename[1]).find(".col") != string::npos)
      ReadColFile(filename[1], &graph, &V);
   else if (string(filename[1]).find(".mm") != string::npos) 
      ReadMMFile(filename[1], &graph, &V);
   else
      

}

/*
  Generate random list based on probabilities
*/
__global__
void Graph_Coloring_GPU(bool* graph,int **color, int V){
  
}

int main(void){
    int **color;
   GraphColoringGPU(argv[1],color);
   /*if (string(argv[1]).find(".col") != string::npos)
      ReadColFile(argv[1], &graph, &V);
   else if (string(argv[1]).find(".mm") != string::npos) 
      ReadMMFile(argv[1], &graph, &V);
   else
      return -1;*/

  return 0;
}